
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <string>
#include <stdlib.h>
#include <math.h>
#include <vector>

using namespace std;

#define N_IN 4
#define N_REC 8
#define N_WINDOW 20
#define TIMESTEPS 20

#define THRESHOLD 0.3
#define VOLT_TAU 20.0
#define VOLT_COEFF exp(-1.0/VOLT_TAU)
#define REF_PERIOD 2

#define W_IN_SIZE N_IN*N_REC
#define W_REC_SIZE N_REC*N_REC
#define NET_STATE_SIZE N_WINDOW*N_REC

float uniform(float low, float high) {
    return low + (static_cast<float>(rand())/RAND_MAX)*(high - low);
}

__global__ void network_step(float voltages[NET_STATE_SIZE],
                             float spike_trains[NET_STATE_SIZE],
                             int refractory_periods[NET_STATE_SIZE],
                             float in_currents[N_WINDOW*N_IN],
                             float weight_in[W_IN_SIZE],
                             float weight_rec[W_REC_SIZE],
                             int t) {

    int pre = blockIdx.x*blockDim.x + threadIdx.x;
    int post = blockIdx.y*blockDim.y + threadIdx.y;

    int last_t = (t - 1)%N_WINDOW;

    //compute recurrent synapses
    __shared__ float rec_synapses[W_REC_SIZE];
    if(pre < N_REC && post < N_REC) {
        float last_spike = spike_trains[N_REC*last_t + pre];
        rec_synapses[N_REC*pre + post] = last_spike*weight_rec[N_REC*pre + post];
    }
    __syncthreads();

    //compute input synapses
    __shared__ float in_synapses[W_IN_SIZE];
    if(pre < N_IN && post < N_REC) {
        int tn = t%N_WINDOW;
        float current_current = in_currents[N_IN*tn + pre];
        in_synapses[N_REC*pre + post] = current_current*weight_in[N_REC*pre + post];
    }
    __syncthreads();

    //printf("%d  %d  %f\n", row, col, elmntwise_mul[N_REC*row + col]);

    //compute new voltages and spikes
    if(pre == 0 && post < N_REC) {

        int tm = t % N_WINDOW;
        int index = N_REC*tm + post;
        int last_index = N_REC*last_t + post;

        //if a spike occurred in the last step, or we are in the refractory period, clamp the voltage and spike trains to 0
        if(spike_trains[last_index] > 0.5 || refractory_periods[last_index] > 0) {
            voltages[index] = 0.0;
            spike_trains[index] = 0.0;
            refractory_periods[index] = (1 + refractory_periods[N_REC*last_t + post])%REF_PERIOD;
        }

        //otherwise sum the synaptic potentials and possibly generate a spike
        else {

            voltages[index] = VOLT_COEFF*voltages[N_REC*last_t + post];

            for(size_t pre_ = 0; pre_ < N_REC; ++pre_) {
                voltages[index] += rec_synapses[N_REC*pre_ + post];
            }

            for(size_t pre_ = 0; pre_ < N_IN; ++pre_) {
                voltages[index] += in_synapses[N_REC*pre_ + post];
            }

            spike_trains[index] = voltages[index] > THRESHOLD ? 1.0 : 0.0;
        }
    }
}

__global__ void stepLIF(float* voltages,
                        float* spike_trains,
                        int* refractory_buffer,
                        float* in_currents,
                        float* weights_in,
                        float* weights_rec,
                        int t) {

    int post = blockIdx.x*blockDim.x + threadIdx.x;

    //compute new voltages and spikes
    if(post < N_REC) {

        int last_t = (t - 1)%N_WINDOW;
        int tm = t%N_WINDOW;
        int index = N_REC*tm + post;
        int last_index = N_REC*last_t + post;

        //if a spike occurred in the last step, or we are in the refractory period, clamp the voltage and spike trains to 0
        if(spike_trains[last_index] > 0.5 || refractory_buffer[last_index] > 0) {
            voltages[index] = 0.0;
            spike_trains[index] = 0.0;
            refractory_buffer[index] = (1 + refractory_buffer[last_index])%REF_PERIOD;
        }

        //otherwise sum the synaptic potentials and possibly generate a spike
        else {

            voltages[index] = VOLT_COEFF*voltages[last_index];

            //recurrent
            for(size_t pre_ = 0; pre_ < N_REC; ++pre_) {
                voltages[index] += weights_rec[N_REC*pre_ + post]*spike_trains[N_REC*last_t + pre_]; 
            }

            //input
            for(size_t pre_ = 0; pre_ < N_IN; ++pre_) {
                voltages[index] += weights_in[N_REC*pre_ + post]*in_currents[N_IN*last_t + pre_];
            }

            spike_trains[index] = voltages[index] > THRESHOLD ? 1.0 : 0.0;
        }
    }
}

int main() {

    float w_rec[W_REC_SIZE];
    for(size_t i = 0; i < N_REC; ++i) {
        for(size_t j = 0; j < N_REC; ++j) {
            w_rec[N_REC*i + j] = uniform(-1.0, 1.0);
        }
    //printf("\n");
    }
    
    float w_in[W_IN_SIZE];
    for(size_t i = 0; i < N_IN; ++i) {
        for(size_t j = 0; j < N_REC; ++j) {
            w_in[N_REC*i + j] = uniform(0.0, 1.0);
        }
    //printf("\n");
    }

    float volts[NET_STATE_SIZE];
    for(size_t i = 0; i < N_WINDOW; ++i) {
        for(size_t j = 0; j < N_REC; ++j) {
            volts[N_REC*i + j] = uniform(-1.0, 1.0);
        }
    }

    float spikes[NET_STATE_SIZE];
    for(size_t i = 0; i < N_WINDOW; ++i) {
        for(size_t j = 0; j < N_REC; ++j) {
            spikes[N_REC*i + j] = 0.0;
        }
    }

    float in_currents[N_WINDOW*N_IN];
    for(size_t i = 0; i < N_WINDOW; ++i) {
        for(size_t j = 0; j < N_REC; ++j) {
            in_currents[N_REC*i + j] = uniform(0.0, 0.3);
        }
    }

    int ref_periods[NET_STATE_SIZE];
    for(size_t i = 0; i < N_WINDOW; ++i) {
        for(size_t j = 0; j < N_REC; ++j) {
            ref_periods[N_REC*i + j] = 0;
        }
    }

    //printf("\n");
    float* w_in_gpu;
    float* w_rec_gpu;
    float* volts_gpu;
    float* spikes_gpu;
    int* ref_periods_gpu;
    float* in_currents_gpu; //pre-determined current so that we don't have to genereate it every time step

    hipMalloc(&w_in_gpu, W_IN_SIZE*sizeof(float));
    hipMalloc(&w_rec_gpu, W_REC_SIZE*sizeof(float));
    hipMalloc(&volts_gpu, NET_STATE_SIZE*sizeof(float));
    hipMalloc(&spikes_gpu, NET_STATE_SIZE*sizeof(float));
    hipMalloc(&ref_periods_gpu, NET_STATE_SIZE*sizeof(int));
    hipMalloc(&in_currents_gpu, N_IN*N_WINDOW*sizeof(float));

    hipMemcpy(w_in_gpu, w_in, W_IN_SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(w_rec_gpu, w_rec, W_REC_SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(volts_gpu, volts, NET_STATE_SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(spikes_gpu, spikes, NET_STATE_SIZE*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(ref_periods_gpu, ref_periods, NET_STATE_SIZE*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(in_currents_gpu, in_currents, N_WINDOW*N_IN*sizeof(float), hipMemcpyHostToDevice);

    for(int time = 1; time < TIMESTEPS; ++time) {

        
        if(time%N_WINDOW == 0) {
            for(size_t i = 0; i < N_WINDOW; ++i) {
                for(size_t j = 0; j < N_REC; ++j) {
                    in_currents[N_REC*i + j] = 0.0;//uniform(0.0, 1.0);
                }
            }
            hipMemcpy(in_currents_gpu, in_currents, N_WINDOW*N_IN*sizeof(float), hipMemcpyHostToDevice);
        }
        

        stepLIF<<<4, N_REC/3>>>(volts_gpu, spikes_gpu, ref_periods_gpu, in_currents_gpu, w_in_gpu, w_rec_gpu, time);
    }

    hipMemcpy(w_in, w_in_gpu, W_IN_SIZE*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(w_rec, w_rec_gpu, W_REC_SIZE*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(volts, volts_gpu, NET_STATE_SIZE*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(spikes, spikes_gpu, NET_STATE_SIZE*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(ref_periods, ref_periods_gpu, NET_STATE_SIZE*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(w_in_gpu);
    hipFree(w_rec_gpu);
    hipFree(volts_gpu);
    hipFree(spikes_gpu);
    hipFree(ref_periods_gpu);
    hipFree(in_currents_gpu);

    for(size_t i = 0; i < N_WINDOW; ++i) {
        //printf("%2d", i);
        cout << setw(5) << i;
        for(size_t j = 0; j < N_REC; ++j) {
            //printf("  ");
            //printf("%4.2f", volts[N_REC*i + j]);
            cout << "  ";
            cout << fixed << setprecision(2) << setw(5) << volts[N_REC*i + j];
        }
        //printf("    ");
        cout << "    ";
        for(size_t j = 0; j < N_REC; ++j) {
            //printf("  ");
            //printf("%4.2f", spikes[N_REC*i + j]);
            cout << "  ";
            cout << fixed << setprecision(2) << setw(5) << spikes[N_REC*i + j];
        }
        cout << endl;
    }

    return 0;

}
